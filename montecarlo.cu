//
//  montecarlo.cu
//  MonteCarloOptions
//
//  Created by Isaac Drachman on 1/15/15.
//  Copyright (c) 2015 Isaac Drachman. All rights reserved.
//

#include <cstdlib>
#include <iostream>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// define struct for passing simulation parameters
typedef struct
{
    int N;
    double S0;
    double sigma;
    double r;
    double T;
    double K;
} params;

/*
 description:   initializes curand (CUDA rng)
 parameters:    
                curandState *state:    pointer to the random number generator
                unsigned int *seed:    value with which to seed rngs
 output:        none       
*/
__global__ void init_curand(hiprandState *state, unsigned int *seed) 
{
    // we seed an rng for each trajectory
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(*seed, idx, 0, &state[idx]);
}

/*
 description:   runs a single trajectory in the monte carlo simulation
 parameters:    
                curandState *state:    pointer to the random number generator
                params *p:             parameters for simulation
                double *payoffs:       payoff array to put result
 output:        none       
*/
__global__ void single_trajectory(hiprandState *state, params *p, double *payoffs) 
{
    // calculate our index (which trajectory is this)
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // set stepsize and initial price
    double dt = p->T / p->N;
    double S = p->S0;
    // for each time step 1 through N
    for (int n = 1; n <= p->N; n++)
    {
        // using Euler-Murayama discretization for the geometric Bronwian model of stock dynamics
        // drift and diffuse the stock price over one timestep
        // taking advantage of curand's standard normal distribution draw function
        S *= (1 + p->r*dt + p->sigma*hiprand_normal(&state[idx])*sqrt(dt));
    }
    // if this option makes money, we have a payoff, otherwise 0
    if (S - p->K > 0.0) payoffs[idx] = S - p->K;
    else payoffs[idx] = 0.0;
}

int main(int argc, char **argv) 
{
    // default number of blocks is 200
    // each block runs 1024 threads (trajectories)
    int nBlocks = 200;
    int nThreads = 1024;

    // check for command line arguments
    if (argc == 3 && strcmp(argv[1],"-b") == 0) {
        // set custom number of blocks
        nBlocks = atoi(argv[2]);
    } else if (argc > 1) {
        // usage error
        std::cout << "usage: " << argv[0] << " [-b blocks]" << std::endl;
        return -1;
    }

    // fill out simulation parameters to pass to GPU
    params h_params;
    h_params.N = 500;            // timesteps
    h_params.S0 = 1992.67;       // spot price
    h_params.sigma = 0.17056;    // volatility (annualized)
    h_params.r = 0.00023;        // risk-free interest rate (annualized)
    h_params.T = 9.0/365.0;      // time to maturity in years
    h_params.K = 1990.00;        // strike price

    // seed the random number generator
    // I'm using time on the host system and passing it to the GPU
    unsigned int h_seed = (unsigned int)time(NULL);
    // make space in VRAM and copy over seed
    unsigned int *d_seed;
    hipMalloc(&d_seed, sizeof(unsigned int));
    hipMemcpy(&d_seed, &h_seed, sizeof(unsigned int), hipMemcpyHostToDevice);

    // setup rng
    hiprandState *d_state;
    // we'll have a seperate generator for each trajectory
    hipMalloc(&d_state, nBlocks * nThreads);
    // initialize this generator with seed
    init_curand<<< nBlocks, nThreads >>>(d_state, d_seed);

    // make space in VRAM and copy over parameters
    params *d_params;
    hipMalloc(&d_params, sizeof(params));
    hipMemcpy(d_params, &h_params, sizeof(params), hipMemcpyHostToDevice);

    // make space on device for payoff array
    double *d_payoffs;
    hipMalloc(&d_payoffs, sizeof(double) * nBlocks * nThreads);

    // run our trajectories
    single_trajectory<<< nBlocks, nThreads >>>(d_state, d_params, d_payoffs);

    // dynamically allocate payoff array on host
    double *h_payoffs = new double[nBlocks*nThreads];
    // copy payoffs from device to host
    hipMemcpy(h_payoffs, d_payoffs, sizeof(double) * nBlocks * nThreads, hipMemcpyDeviceToHost);

    // sum each payoff
    double sum = 0.0;
    for (int m = 0; m < nBlocks*nThreads; m++) sum += h_payoffs[m];    

    // calculate discounted average payoff
    double premium = exp(-h_params.r * h_params.T)*(sum/(nBlocks*nThreads));
    // print result
    std::cout << "european call premium = " << premium << std::endl;

    // free memory on host and device
    delete[] h_payoffs;
    hipFree(d_state);
    hipFree(d_params);
    hipFree(d_payoffs);

    // exit
    return 0;
}